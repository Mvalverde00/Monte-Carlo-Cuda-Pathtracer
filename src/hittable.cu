#include "hip/hip_runtime.h"
﻿#include "hittable.cuh"

#include <glm/gtx/norm.hpp>


Sphere::Sphere() : center(glm::vec3(0.0, 0.0, 0.0)), radius(1.0f), matIdx(-1) {};
Sphere::Sphere(glm::vec3 center, float radius, int matIdx) : center(center), radius(radius), matIdx(matIdx) {};

CUDA_CALLABLE_MEMBER void Sphere::hit(const Ray& r, float t_min, float t_max, HitRecord& rec) {
  // Math taken from https://en.wikipedia.org/wiki/Line%E2%80%93sphere_intersection

  glm::vec3 oc = r.origin - center;

  float a = glm::length2(r.dir);
  float b = 2.0 * glm::dot(r.dir, oc);
  float c = glm::length2(oc) - radius * radius;

  float discriminant = b * b - 4.0 * a * c;
  if (discriminant < 0.0) {
    rec.isHit = false;
    return;
  }
  float sqrtd = std::sqrt(discriminant);

  float root = (-b - sqrtd) / (2.0 * a);
  if (root < t_min || root > t_max) {
    root = (-b + sqrtd) / (2.0 * a);
    if (root < t_min || root > t_max) {
      rec.isHit = false;
      return;
    }
  }

  rec.t = root;
  rec.point = r.at(root);
  rec.setNormal(r, (rec.point - center) / radius);
  //rec.normal = (rec.point - center) / radius;
  rec.isHit = true;
  rec.matIdx = matIdx;
}

CUDA_CALLABLE_MEMBER glm::vec3 Triangle::getNormal(float u, float v) {
  return (1.0f - u - v) * v0.n + u * v1.n + v * v2.n;
}

CUDA_CALLABLE_MEMBER void Triangle::hit(const Ray& r, float t_min, float t_max, HitRecord& rec) {
  glm::vec3 e0 = v1.pos - v0.pos;
  glm::vec3 e1 = v2.pos - v0.pos;
  glm::vec3 h = glm::cross(r.dir, e1);
  float a = glm::dot(e0, h);

  if (a > -0.00001 && a < 0.00001) {
    rec.isHit = false;
    return;
  }

  float f = 1.0 / a;
  glm::vec3 s = r.origin - v0.pos;
  float u = f * glm::dot(s, h);
  if (u < 0.0 || u > 1.0) {
    rec.isHit = false;
    return;
  }

  glm::vec3 q = glm::cross(s, e0);
  float v = f * glm::dot(r.dir, q);
  if (v < 0.0 || u + v > 1.0) {
    rec.isHit = false;
    return;
  }

  float t = f * glm::dot(e1, q);
  if (t > t_min && t < t_max) {
    rec.point = r.origin + t * r.dir;
    rec.t = t;
    rec.u = u;
    rec.v = v;
    //rec.setNormal(r, getNormal(u, v));
    rec.normal = getNormal(u, v);
    rec.isHit = true;
    return;
  }

  rec.isHit = false;
}