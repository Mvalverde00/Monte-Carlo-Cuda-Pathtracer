#include "hip/hip_runtime.h"
﻿#include "pathtrace.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include "hip/hip_runtime_api.h"
#include "camera.cuh"
#include "hittable.cuh"
#include "material.cuh"
#include "mesh.cuh"

surface<void, cudaSurfaceType2D> surf;


PTData::PTData(hiprandState* d_rand, glm::vec3* d_accum, Camera* d_cam) {
  rand = d_rand;
  accum = d_accum;
  cam = d_cam;

  samples = 0;
  renderTime = 0.0f;
  reset = false;
  showNormals = false;

  n_sphs = n_meshes = 0;
  sph = NULL;
  tris = NULL;
  meshes = NULL;
  mats = NULL;
}



__device__ void intersectSpheres(const Ray& r, float t_min, float t_max, HitRecord& hr, Sphere* sphs, int n_sphs) {
  HitRecord temp;
  temp.isHit = hr.isHit = false;
  hr.t = t_max;

  for (int i = 0; i < n_sphs; i++) {
    sphs[i].hit(r, t_min, t_max, temp);

    if (temp.isHit && temp.t < hr.t) {
      hr = temp;
    }
  }
}


__device__ void intersectMeshes(const Ray& r, float t_min, float t_max, HitRecord& hr, PTData args) {
  HitRecord temp;
  temp.isHit = hr.isHit = false;
  hr.t = t_max;

  for (int i = 0; i < (args).n_meshes; i++) {
    args.meshes[i].hit(r, t_min, t_max, temp, args.tris);
    temp.matIdx = args.meshes[i].matIdx;
    if (temp.isHit && temp.t < hr.t) {
      hr = temp;
    }
  }
}

__device__ void intersectScene(const Ray& r, float t_min, float t_max, HitRecord& hr, PTData args) {
  HitRecord sph, mesh;
  intersectSpheres(r, t_min, t_max, sph, args.sph, args.n_sphs);
  intersectMeshes(r, t_min, t_max, mesh, args);

  if (sph.t < mesh.t)
    hr = sph;
  else
    hr = mesh;
}

__device__ glm::vec3 rayColor(Ray& r, HitRecord& hr, PTData& args, hiprandState* rand) {
  glm::vec3 accumulate = glm::vec3(1, 1, 1);
  for (int i = 0; i < 10; i++) {
    //intersectSpheres(r, 0.001f, 999999.0f, hr, args.sph, args.n_sphs);
    intersectScene(r, 0.001f, 999999.0f, hr, args);

    if (hr.isHit) {
      if (args.showNormals)
        return (glm::normalize(hr.normal) + glm::vec3(1.0, 1.0, 1.0)) / 2.0f;

      glm::vec3 attenuation;
      if (args.mats[hr.matIdx].scatter(r, hr, attenuation, rand)) {
        accumulate *= attenuation;
      }

    } else {
      // Ray shoots off into background
      return accumulate * glm::vec3(1, 1, 1);
    }
  }

  // Ray got stuck, never escaped.
  return glm::vec3(0, 0, 0);
}

__global__ void writeColors(unsigned int width, unsigned int height, PTData args) {
  unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
  unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;

  if (x < width && y < height) {
    Ray r = args.cam->getRay(x, y, width, height, &args.rand[y * width + x]);
    r = Ray(r.at(1.0f), r.dir);
    HitRecord hr;

    glm::vec3 sample_value = rayColor(r, hr, args, &args.rand[y * width + x]);


    if (args.reset) {
      args.accum[y * width + x] = sample_value;
    } else {
      args.accum[y * width + x] += sample_value;
    }
    glm::vec3 res = args.accum[y * width + x] / float(args.samples);
    uchar4 color = make_uchar4(255 * res.x, 255 * res.y, 255 * res.z, 255);
    surf2Dwrite(color, surf, x * sizeof(color), y, hipBoundaryModeZero);
  }
}

void drawToScreen(int XRES, int YRES, hipArray_const_t array, PTData& args) {
  CUDA_CALL(cudaBindSurfaceToArray(surf, array));
  const int blockX = 16;
  const int blockY = 16;
  dim3 blockSize(blockX, blockY);
  dim3 gridSize((XRES+ blockX - 1) / blockX, (YRES+ blockY - 1) / blockY);
  writeColors<<<gridSize, blockSize>>>((unsigned int)XRES, (unsigned int)YRES, args);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
}







__global__ void cudaInitRandom(int width, int height, hiprandState* d_rand) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < width && y < height) {
    int idx = y * width + x;
    hiprand_init(2021, idx, 0, &d_rand[idx]);
  }
}


void initRandom(int XRES, int YRES, hiprandState* d_curand_state) {
  const int blockX = 32;
  const int blockY = 32;
  dim3 blockSize(blockX, blockY);
  dim3 gridSize((XRES + blockX - 1) / blockX, (YRES + blockY - 1) / blockY);
  cudaInitRandom<<<gridSize, blockSize>>>(XRES, YRES, d_curand_state);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
}